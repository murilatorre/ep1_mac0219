#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0

#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10


__global__ void jacobi_iteration(double *h, double *g, int n, int iter_limit)
{
    for (int iter = 0; iter < iter_limit; iter++) {
        for (int i = 1; i < n - 1; i++)
            for (int j = 1; j < n - 1; j++)
                g[i*n + j] = 0.25 * (h[(i-1)*n + j] + h[(i+1)*n + j] + h[i*n + (j-1)] + h[i*n + (j+1)]);
            
    
        for (int i = 1; i < n - 1; i++)
            for (int j = 1; j < n - 1; j++)
                h[i*n + j] = g[i*n + j];
    }
}

void c_jacobi_iteration(double *h, double *g, int n, int iter_limit)
{
    for (int iter = 0; iter < iter_limit; iter++) {
        for (int i = 1; i < n - 1; i++)
            for (int j = 1; j < n - 1; j++)
                g[i*n + j] = 0.25 * (h[(i-1)*n + j] + h[(i+1)*n + j] + h[i*n + (j-1)] + h[i*n + (j+1)]);
            
    
        for (int i = 1; i < n - 1; i++)
            for (int j = 1; j < n - 1; j++)
                h[i*n + j] = g[i*n + j];
    }
}

void initialize(double *h, int n)
{
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1)
                h[i*n + j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            else
                h[i*n + j] = 0.0;
        }
    }
}

double calculate_elapsed_time(struct timespec start, struct timespec end)
{
    double start_sec = (double)start.tv_sec * 1e9 + (double)start.tv_nsec;
    double end_sec = (double)end.tv_sec * 1e9 + (double)end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

void save_to_file(double *h, int n)
{
    FILE *file = fopen("room.txt", "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            fprintf(file, "%lf ", h[i*n + j]);
        fprintf(file, "\n");
    }
    fclose(file);
}

int main(int argc, char *argv[])
{
    if (argc < 3) {
        fprintf(stderr, "Uso: %s <número de pontos> <limite de iterações>\n", argv[0]);
        return 1;
    }

    struct timespec start_host, end_host;
    struct timespec start_device, end_device;
    double *h, *g;
    double *d_h, *d_g; 

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);

    h = (double *)malloc(n*n * sizeof(double));
    g = (double *)malloc(n*n * sizeof(double));
    if (h == NULL || g == NULL) {
        fprintf(stderr, "Erro ao alocar memória para h ou g\n");
        exit(EXIT_FAILURE);
    }

    initialize(h, n);

    // Allocate device memory 
    hipMalloc((void**)&d_h, n*n * sizeof(double));
    hipMalloc((void**)&d_g, n*n * sizeof(double));

    // Transfer data from host to device memory
    hipMemcpy(d_h, h, n*n * sizeof(double), hipMemcpyHostToDevice);

    // Executing kernel 
    int block_size = 256;
    int grid_size = ((n + block_size) / block_size);

    clock_gettime(CLOCK_MONOTONIC, &start_device);
    jacobi_iteration<<<1,1>>>(d_h, d_g, n, iter_limit);
    clock_gettime(CLOCK_MONOTONIC, &end_device);
    
    // Transfer data back to host memory
    hipMemcpy(h, d_h, n*n * sizeof(double), hipMemcpyDeviceToHost);
    save_to_file(h, n);

    // Verification

    // printf("Tempo de execução CPU: %.9f segundos\n", calculate_elapsed_time(start_host, end_host));
    printf("Tempo de execução GPU: %.9f segundos\n", calculate_elapsed_time(start_device, nd_device));

    // Deallocate device memory
    hipFree(d_h);
    hipFree(d_g);

    // Deallocate host memory
    free(h); 
    free(g); 

    return 0;
}